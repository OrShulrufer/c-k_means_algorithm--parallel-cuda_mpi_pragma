#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "StructsHeader.h"

#define THREADS_RER_BLOCK 1000


__global__ void movePoints (Point*devPoints, int allPointsSize, unsigned int numofThreadsperBlock, unsigned int numofBlocks, double theTime, int flag) {
	int index = blockIdx.x * numofThreadsperBlock + threadIdx.x;
	/*Amount of Work per Thread */
	int threadwork = allPointsSize /(numofThreadsperBlock*(numofBlocks-flag));

	for (int i = index*threadwork; i < (index* threadwork) + threadwork; i++)
	{
	if (i < allPointsSize) {
		    devPoints[i].x  = devPoints[i].x + theTime*devPoints[i].Vx;
			devPoints[i].y = devPoints[i].y + theTime*devPoints[i].Vy;
			devPoints[i].z = devPoints[i].z + theTime*devPoints[i].Vz;
		}
	}
}

hipError_t pointsLocation(int allPointsSize, double theTime, Point* pointsArray) {

	Point* devPoints = NULL;

	unsigned int numofThreadsperBlock = THREADS_RER_BLOCK;
	unsigned int numofBlocks = allPointsSize / numofThreadsperBlock;

	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		hipFree(devPoints);
	}
	cudaStatus = hipMalloc((void**)&devPoints, allPointsSize * sizeof(Point));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		hipFree(devPoints);
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(devPoints, pointsArray, allPointsSize * sizeof(Point), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		hipFree(devPoints);
	}

	int flag = 0;
	if (0 < allPointsSize % (numofThreadsperBlock)) {
	    numofBlocks += 1;
		flag = 1;
    }
		// Launch a kernel on the GPU
		movePoints << <numofBlocks, numofThreadsperBlock >> > (devPoints,  allPointsSize, numofThreadsperBlock, numofBlocks, theTime, flag);
	

		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "CalculateMaximumClusterDiameters launch failed: %s\n", hipGetErrorString(cudaStatus));
		hipFree(devPoints);
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		hipFree(devPoints);
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(pointsArray, devPoints, allPointsSize * sizeof(Point), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		hipFree(devPoints);
	}

	hipFree(devPoints);
	return cudaStatus;
}

